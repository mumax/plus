#include <iomanip>
#include <iostream>
#include <map>
#include <utility>

#include "cudaerror.hpp"
#include "gpumemorypool.hpp"

GpuMemoryPool memoryPool;

GpuMemoryPool::~GpuMemoryPool() {
  for (const auto& poolEntry : pool_)
    for (auto& ptr : poolEntry.second)
      checkCudaError(hipFree(ptr));
}

void* GpuMemoryPool::allocate(size_t size) {
  void* ptr;
  if (pool_[size].size() == 0) {
    checkCudaError(hipMalloc(reinterpret_cast<void**>(&ptr), size));
  } else {
    ptr = pool_[size].back();
    pool_[size].pop_back();
  }
  inUse_[ptr] = size;
  return ptr;
}

void GpuMemoryPool::free(void** ptr) {
  inUse_.erase(*ptr);
  checkCudaError(hipFree(*ptr));
  *ptr = nullptr;
}

void GpuMemoryPool::recycle(void** ptr) {
  auto inUseIt = inUse_.find(*ptr);
  int size = inUseIt->second;
  inUse_.erase(inUseIt);
  pool_[size].push_back(*ptr);
  *ptr = nullptr;
}

void GpuMemoryPool::printInfo() const {
  // memoryUse map:
  //   key           memory block size
  //   value.first   in use count
  //   value.second  in pool count
  std::map<size_t, std::pair<int, int>> memoryUse;

  // count number of used memory blocks (for each memory block size seperately)
  for (auto u : inUse_)
    memoryUse[u.second].first++;

  // get number of memory blocks in the pool for each memory block size
  for (const auto& p : pool_)
    memoryUse[p.first].second = p.second.size();

  int totalMemUsed = 0;
  for (auto b : memoryUse) {
    totalMemUsed += b.first * (b.second.first + b.second.second);
  }

  std::cout << "GPU MEMORY POOL INFORMATION:" << std::endl;
  int colwidth = 10;
  std::cout << std::setw(colwidth) << "size(bytes)";
  std::cout << std::setw(colwidth) << "in use";
  std::cout << std::setw(colwidth) << "free" << std::endl;
  for (auto b : memoryUse) {
    std::cout << std::setw(colwidth) << b.first;
    std::cout << std::setw(colwidth) << b.second.first;
    std::cout << std::setw(colwidth) << b.second.second << std::endl;
  }
  std::cout << "Total used GPU memory:  " << totalMemUsed << " bytes"
            << std::endl;
}
