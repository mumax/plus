#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include <memory>
#include <vector>

#include "antiferromagnet.hpp"
#include "constants.hpp"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "fieldops.hpp"
#include "magnet.hpp"
#include "quantityevaluator.hpp"
#include "field.hpp"
#include "grid.hpp"
#include "parameter.hpp"
#include "strayfieldfft.hpp"
#include "strayfieldkernel.hpp"
#include "system.hpp"

#if FP_PRECISION == SINGLE
const hipfftType FFT = HIPFFT_R2C;
const hipfftType IFFT = HIPFFT_C2R;
const auto& fftExec = hipfftExecR2C;
const auto& ifftExec = hipfftExecC2R;
#elif FP_PRECISION == DOUBLE
const hipfftType FFT = HIPFFT_D2Z;
const hipfftType IFFT = HIPFFT_Z2D;
const auto& fftExec = hipfftExecD2Z;
const auto& ifftExec = hipfftExecZ2D;
#endif

#define __CUDAOP__ inline __device__ __host__

// No simpel operator overloading due to definition of real2.
__CUDAOP__ complex sum(complex a, complex b) {
#if FP_PRECISION == SINGLE
  return hipCaddf(a, b);
#elif FP_PRECISION == DOUBLE
  return hipCadd(a, b);
#endif
}

__CUDAOP__ complex prod(complex a, complex b) {
#if FP_PRECISION == SINGLE
  return hipCmulf(a, b);
#elif FP_PRECISION == DOUBLE
  return hipCmul(a, b);
#endif
}

__global__ void k_pad(CuField out, CuField in1, CuParameter msat1, CuField in2, CuParameter msat2, real fac) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;
  
  Grid outgrid = out.system.grid;
  Grid ingrid = in1.system.grid;

  if (outIdx >= outgrid.ncells())
    return;

  int3 outCoo = outgrid.index2coord(outIdx);
  int3 inCoo = outCoo - outgrid.origin() + ingrid.origin();
  int inIdx = ingrid.coord2index(inCoo);

  if (in1.cellInGeometry(inCoo)) {
    real Ms1 = msat1.valueAt(inIdx);
    real Ms2 = msat2.valueAt(inIdx);
    for (int c = 0; c < out.ncomp; c++)
      out.setValueInCell(outIdx, c, (Ms1 * in1.valueAt(inIdx, c) + Ms2 * in2.valueAt(inIdx, c)) / fac);
  } else {
    for (int c = 0; c < out.ncomp; c++)
      out.setValueInCell(outIdx, c, 0.0);
  }
}

__global__ void k_unpad(CuField out, CuField in) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  // When outside the geometry of destiny field, set to zero and return
  // early
  if (!out.cellInGeometry(outIdx)) {
    if (out.cellInGrid(outIdx))
        out.setVectorInCell(outIdx, real3{0, 0, 0});
    return;
  }

  Grid outgrid = out.system.grid;
  Grid ingrid = in.system.grid;

  // Output coordinate relative to the origin of the output grid
  int3 outRelCoo = outgrid.index2coord(outIdx) - outgrid.origin();

  // Input coordinate relative to the origin of the input grid
  int3 inRelCoo = ingrid.size() - outgrid.size() + outRelCoo;

  int inIdx = ingrid.coord2index(inRelCoo + ingrid.origin());

  for (int c = 0; c < out.ncomp; c++) {
    out.setValueInCell(outIdx, c, in.valueAt(inIdx, c));
  }
}

static void checkCufftResult(hipfftResult result) {
  if (result != HIPFFT_SUCCESS)
    throw std::runtime_error("cufft error in demag convolution");
}

__global__ void k_apply_kernel_3d(complex* hx,
                                  complex* hy,
                                  complex* hz,
                                  complex* mx,
                                  complex* my,
                                  complex* mz,
                                  complex* kxx,
                                  complex* kyy,
                                  complex* kzz,
                                  complex* kxy,
                                  complex* kxz,
                                  complex* kyz,
                                  complex preFactor,
                                  int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  hx[i] = prod(preFactor, (sum(sum(prod(kxx[i], mx[i]), prod(kxy[i], my[i])), prod(kxz[i], mz[i]))));
  hy[i] = prod(preFactor, (sum(sum(prod(kxy[i], mx[i]), prod(kyy[i], my[i])), prod(kyz[i], mz[i]))));
  hz[i] = prod(preFactor, (sum(sum(prod(kxz[i], mx[i]), prod(kyz[i], my[i])), prod(kzz[i], mz[i]))));
}

__global__ void k_apply_kernel_2d(complex* hx,
                                  complex* hy,
                                  complex* hz,
                                  complex* mx,
                                  complex* my,
                                  complex* mz,
                                  complex* kxx,
                                  complex* kyy,
                                  complex* kzz,
                                  complex* kxy,
                                  complex preFactor,
                                  int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  hx[i] = prod(preFactor, (sum(prod(kxx[i], mx[i]), prod(kxy[i], my[i]))));
  hy[i] = prod(preFactor, (sum(prod(kxy[i], mx[i]), prod(kyy[i], my[i]))));
  hz[i] = prod(preFactor, prod(kzz[i], mz[i]));
}

StrayFieldFFTExecutor::StrayFieldFFTExecutor(
    const Magnet* magnet,
    std::shared_ptr<const System> system, int order, double switchingradius)
    : StrayFieldExecutor(magnet, system),
      kernel_(system->grid(), magnet_->grid(), magnet->world(), order, switchingradius),
      kfft(6),
      hfft(3),
      mfft(3) {
  int3 size = kernel_.grid().size();
  fftSize = {size.x / 2 + 1, size.y, size.z};
  int ncells = fftSize.x * fftSize.y * fftSize.z;

  for (auto& p : kfft)
    hipMalloc(reinterpret_cast<void**>(&p), ncells * sizeof(complex));
  for (auto& p : mfft)
    hipMalloc(reinterpret_cast<void**>(&p), ncells * sizeof(complex));
  for (auto& p : hfft)
    hipMalloc(reinterpret_cast<void**>(&p), ncells * sizeof(complex));

  checkCufftResult(hipfftPlan3d(&forwardPlan, size.z, size.y, size.x, FFT));
  checkCufftResult(hipfftPlan3d(&backwardPlan, size.z, size.y, size.x, IFFT));

  hipfftSetStream(forwardPlan, getCudaStream());
  hipfftSetStream(backwardPlan, getCudaStream());

  for (int comp = 0; comp < 6; comp++)
    checkCufftResult(
        fftExec(forwardPlan, kernel_.field().device_ptr(comp), kfft.at(comp)));
}

StrayFieldFFTExecutor::~StrayFieldFFTExecutor() {
  for (auto p : mfft)
    hipFree(p);
  for (auto p : kfft)
    hipFree(p);
  for (auto p : hfft)
    hipFree(p);
  
  checkCufftResult(hipfftDestroy(forwardPlan));
  checkCufftResult(hipfftDestroy(backwardPlan));
}

Field StrayFieldFFTExecutor::exec() const {

  // pad m, and multiply with msat
  std::shared_ptr<System> kernelSystem =
      std::make_shared<System>(magnet_->world(), kernel_.grid());
  std::unique_ptr<Field> mpad(new Field(kernelSystem, 3));

  // Launch kernel function in different scopes to avoid unnecessary copies
  real fac;
  if (const Ferromagnet* mag = dynamic_cast<const Ferromagnet*>(magnet_)) {
    auto m = mag->magnetization()->field().cu();
    auto ms = mag->msat.cu();
    fac = 2.0;
    hipLaunchByPtr(mpad->grid().ncells(), k_pad, mpad->cu(), m, ms, m, ms, fac);
  }
  else if (const Antiferromagnet* mag = dynamic_cast<const Antiferromagnet*>(magnet_)) {
    auto m1 = mag->sub1()->magnetization()->field().cu();
    auto m2 = mag->sub2()->magnetization()->field().cu();
    auto ms1 = mag->sub1()->msat.cu();
    auto ms2 = mag->sub2()->msat.cu();
    fac = 1.0;
    hipLaunchByPtr(mpad->grid().ncells(), k_pad, mpad->cu(), m1, ms1, m2, ms2, fac);
  }

  // Forward fourier transforms
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
        fftExec(forwardPlan, mpad->device_ptr(comp), mfft.at(comp)));
  
  // apply kernel on m_fft
  int ncells = fftSize.x * fftSize.y * fftSize.z;
  complex preFactor{-MU0 / kernel_.grid().ncells(), 0};
  if (kernel_.grid().size().z == 1 && kernel_.grid().origin().z == 0) {
    // if the h field and m field are two dimensional AND are in the same plane
    // (kernel grid origin at z=0) then the kernel matrix has only 4 relevant
    // components and a more efficient cuda kernel can be used:
    hipLaunchByPtr(ncells, k_apply_kernel_2d, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), preFactor, ncells);
  } else {
    hipLaunchByPtr(ncells, k_apply_kernel_3d, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), kfft.at(4), kfft.at(5), preFactor,
               ncells);
  }

  // backward fourier transfrom
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
      ifftExec(backwardPlan, hfft.at(comp), mpad->device_ptr(comp)));

  // unpad
  Field h(system_, 3);
  hipLaunchByPtr(h.grid().ncells(), k_unpad, h.cu(), mpad->cu());
  return h;
}
