#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "dmi.hpp"
#include "dmitensor.hpp"
#include "energy.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "parameter.hpp"
#include "world.hpp"

bool dmiAssuredZero(const Ferromagnet* magnet) {
  return magnet->dmiTensor.assuredZero() || magnet->msat.assuredZero();
}

__device__ static inline real harmonicMean(real a, real b) {
  if (a + b == 0.0)
    return 0.0;
  return 2 * a * b / (a + b);
}

__device__ static inline real harmonicMean(const CuParameter& param,
                                           int idx1,
                                           int idx2) {
  return harmonicMean(param.valueAt(idx1), param.valueAt(idx2));
}

__global__ void k_dmiField(CuField hField,
                           const CuField mField,
                           const CuDmiTensor dmiTensor,
                           const CuParameter msat,
                           Grid mastergrid) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const auto system = hField.system;

  if (!system.grid.cellInGrid(idx))
    return;

  // When outside the geometry or msat=0, set to zero and return early
  if (!system.inGeometry(idx) || msat.valueAt(idx) == 0) {
    hField.setVectorInCell(idx, {0, 0, 0});
    return;
  }

  const int3 coo = system.grid.index2coord(idx);

  // Accumulate exchange field of cell at idx in h. Divide by msat at the end.
  real3 h{0, 0, 0};

// Loop over the 6 nearest neighbors using the neighbor's relative coordinate.
// Compute for each neighbor the DMI effective field term.
#pragma unroll
  for (int3 relative_coo : {int3{-1, 0, 0}, int3{1, 0, 0}, int3{0, -1, 0},
                            int3{0, 1, 0}, int3{0, 0, -1}, int3{0, 0, 1}}) {
    int3 neighbor_coo = mastergrid.wrap(coo + relative_coo);
    int neighbor_idx = system.grid.coord2index(neighbor_coo);

    // If there is no neighbor, then simply continue without adding to the
    // effective field. By doing this, we assume open boundary conditions.
    if (!system.inGeometry(neighbor_coo) || msat.valueAt(neighbor_idx) == 0)
      continue;

    // Get the dmi strengths between the center cell and the neighbor, which are
    // the harmonic means of the dmi strengths of both cells.
    real Dxz, Dxy, Dyz, Dzx, Dyx, Dzy;
    if (relative_coo.x) {  // derivative along x
      Dxz = harmonicMean(dmiTensor.xxz, idx, neighbor_idx);
      Dxy = harmonicMean(dmiTensor.xxy, idx, neighbor_idx);
      Dyz = harmonicMean(dmiTensor.xyz, idx, neighbor_idx);
    } else if (relative_coo.y) {  // derivative along y
      Dxz = harmonicMean(dmiTensor.yxz, idx, neighbor_idx);
      Dxy = harmonicMean(dmiTensor.yxy, idx, neighbor_idx);
      Dyz = harmonicMean(dmiTensor.yyz, idx, neighbor_idx);
    } else if (relative_coo.z) {  // derivative along z
      Dxz = harmonicMean(dmiTensor.zxz, idx, neighbor_idx);
      Dxy = harmonicMean(dmiTensor.zxy, idx, neighbor_idx);
      Dyz = harmonicMean(dmiTensor.zyz, idx, neighbor_idx);
    }
    Dzx = -Dxz;  // dmi tensor is assymetric
    Dyx = -Dxy;
    Dzy = -Dyz;

    // Distance between neighbors (the finite difference)
    real delta = relative_coo.x * system.cellsize.x +
                 relative_coo.y * system.cellsize.y +
                 relative_coo.z * system.cellsize.z;

    // Compute the effective field contribution of the DMI with the neighbor
    real3 m_ = mField.vectorAt(neighbor_idx);
    h.x += (Dxy * m_.y + Dxz * m_.z) / delta;
    h.y += (Dyx * m_.x + Dyz * m_.z) / delta;
    h.z += (Dzx * m_.x + Dzy * m_.y) / delta;
  }  // end loop over neighbors

  h /= msat.valueAt(idx);
  hField.setVectorInCell(idx, h);
}

Field evalDmiField(const Ferromagnet* magnet) {
  Field hField(magnet->system(), 3);

  if (dmiAssuredZero(magnet)) {
    hField.makeZero();
    return hField;
  }

  hipLaunchByPtr(hField.grid().ncells(), k_dmiField, hField.cu(),
             magnet->magnetization()->field().cu(), magnet->dmiTensor.cu(),
             magnet->msat.cu(), magnet->world()->mastergrid());

  return hField;
}

Field evalDmiEnergyDensity(const Ferromagnet* magnet) {
  if (dmiAssuredZero(magnet))
    return Field(magnet->system(), 1, 0.0);

  return evalEnergyDensity(magnet, evalDmiField(magnet), 0.5);
}

real evalDmiEnergy(const Ferromagnet* magnet) {
  if (dmiAssuredZero(magnet))
    return 0;
  real edens = dmiEnergyDensityQuantity(magnet).average()[0];
  int ncells = magnet->grid().ncells();
  real cellVolume = magnet->world()->cellVolume();
  return ncells * edens * cellVolume;
}

FM_FieldQuantity dmiFieldQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalDmiField, 3, "dmi_field", "T");
}

FM_FieldQuantity dmiEnergyDensityQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalDmiEnergyDensity, 1, "dmi_emergy_density",
                          "J/m3");
}

FM_ScalarQuantity dmiEnergyQuantity(const Ferromagnet* magnet) {
  return FM_ScalarQuantity(magnet, evalDmiEnergy, "dmi_energy", "J");
}
