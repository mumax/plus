#include "hip/hip_runtime.h"
#include <memory>

#include "antiferromagnet.hpp"
#include "constants.hpp"
#include "cudalaunch.hpp"
#include "ferromagnet.hpp"
#include "magnet.hpp"
#include "field.hpp"
#include "fieldops.hpp"
#include "grid.hpp"
#include "parameter.hpp"
#include "strayfieldbrute.hpp"
#include "system.hpp"

__global__ void k_demagfield(CuField hField,
                             const CuField mField1,
                             const CuField mField2,
                             const CuField kernel,
                             const CuParameter msat1,
                             const CuParameter msat2,
                             real fac) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // When outside the geometry of destiny field, set to zero and return
  // early
  if (!hField.cellInGeometry(idx)) {
    if (hField.cellInGrid(idx))
      hField.setVectorInCell(idx, real3{0, 0, 0});
    return;
  }

  int3 dstcoo = hField.system.grid.index2coord(idx);
  real3 h{0, 0, 0};

  for (int i = 0; i < mField1.system.grid.ncells(); i++) {
    if (!mField1.cellInGeometry(i))
      continue;

    int3 srccoo = mField1.system.grid.index2coord(i);
    int3 r = dstcoo - srccoo;
    real nxx = kernel.valueAt(r, 0);
    real nyy = kernel.valueAt(r, 1);
    real nzz = kernel.valueAt(r, 2);
    real nxy = kernel.valueAt(r, 3);
    real nxz = kernel.valueAt(r, 4);
    real nyz = kernel.valueAt(r, 5);
    
    real3 M = (msat1.valueAt(i) * mField1.vectorAt(i) +
               msat2.valueAt(i) * mField2.vectorAt(i)) / fac;

    h.x -= nxx * M.x + nxy * M.y + nxz * M.z;
    h.y -= nxy * M.x + nyy * M.y + nyz * M.z;
    h.z -= nxz * M.x + nyz * M.y + nzz * M.z;
  }
  
  hField.setVectorInCell(idx, MU0 * h);
}

StrayFieldBruteExecutor::StrayFieldBruteExecutor(
    const Magnet* magnet,
    std::shared_ptr<const System> system, int order, double switchingradius)
    : StrayFieldExecutor(magnet, system),
      kernel_(system->grid(), magnet_->grid(), magnet_->world(), order, switchingradius) {}

Field StrayFieldBruteExecutor::exec() const {
  
  Field h(system_, 3);
  int ncells = h.grid().ncells();
  real fac;

  if(const Ferromagnet* mag = dynamic_cast<const Ferromagnet*>(magnet_)) {
    auto m = mag->magnetization()->field().cu();
    auto msat = mag->msat.cu();
    fac = 2.0;
    hipLaunchByPtr(ncells, k_demagfield, h.cu(), m, m, kernel_.field().cu(),
              msat, msat, fac);
  }
  else if (const Antiferromagnet* mag = dynamic_cast<const Antiferromagnet*>(magnet_)) {
    auto m1 = mag->sub1()->magnetization()->field().cu();
    auto m2 = mag->sub2()->magnetization()->field().cu();
    auto ms1 = mag->sub1()->msat.cu();
    auto ms2 = mag->sub2()->msat.cu();
    fac = 1.0;
    hipLaunchByPtr(ncells, k_demagfield, h.cu(), m1, m2, kernel_.field().cu(),
              ms1, ms2, fac);
  }
  return h;
}
