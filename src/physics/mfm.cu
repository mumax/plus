#include "hip/hip_runtime.h"
#include "constants.hpp"
#include "cudalaunch.hpp"
#include "datatypes.hpp"
#include "fullmag.hpp"
#include "mfm.hpp"
#include "system.hpp"
#include <map>
#include <iostream>
#include <stdio.h>

/** This code calculates an MFM kernel
  * Need to calculate dF/dz = sum M . d²B/dz²
  * The sum runs over each cell in a magnet.
  * M is the magnetization in that cell.
  * B is the stray field from the tip evaluated in the cell.
  * Source: The design and verification of MuMax3.
  */
__global__ void k_magneticForceMicroscopy(CuField kernel,
                                          CuField magnetization,
                                          const Grid mastergrid,
                                          const int3 pbcRepetitions,
                                          real lift,
                                          real tipsize,
                                          const real V) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (!kernel.cellInGrid(idx))
        return;

    real pi = 3.1415926535897931;
    
    // The cell-coordinate of the tip (without lift)
    const real3 cellsize = kernel.system.cellsize;
    int3 coo = kernel.system.grid.index2coord(idx);
    real x0 = coo.x * cellsize.x;
    real y0 = coo.y * cellsize.y;
    real z0 = coo.z * cellsize.z;

    real prefactor = 1/(4*pi*MU0);  // charge at the tip is 1/µ0
    real delta = 1e-9;  // tip oscillation, take 2nd derivative over this distance
    // Size of the magnet's grid
    int xmax = magnetization.system.grid.size().x;
    int ymax = magnetization.system.grid.size().y;
    int zmax = magnetization.system.grid.size().z;

    real dFdz = 0.;
    // Loop over valid pbc
    for (int Ny = -pbcRepetitions.y; Ny <= pbcRepetitions.y; Ny++) {
        real ypbc = Ny * mastergrid.size().y;
        for (int Nx = -pbcRepetitions.x; Nx <= pbcRepetitions.x; Nx++) {
            real xpbc = Nx * mastergrid.size().x;            

            // Loop over cells in the magnet
            for (int iz = 0; iz < zmax; iz++) {
                real z = (iz + magnetization.system.grid.origin().z) * cellsize.z;
                for (int iy = 0; iy < ymax; iy++) {
                    real y = (iy + magnetization.system.grid.origin().y + ypbc) * cellsize.y;
                    for (int ix = 0; ix < xmax; ix++) {
                        real x = (ix + magnetization.system.grid.origin().x + xpbc) * cellsize.x;

                        real3 m = magnetization.vectorAt(int3{ix + magnetization.system.grid.origin().x, iy + magnetization.system.grid.origin().y, iz + magnetization.system.grid.origin().z});
                        real E[3];  // Energy of 3 tip positions

                        // Get 3 different tip heights
                        for (int i = -1; i <= 1; i++) {
                            // First pole position and field
                            real3 R = {x0-x,
                                       y0-y,
                                       z0 - z + (lift + i*delta)};
                            real r = norm(R);
                            real3 B = R * prefactor/(r*r*r);
                            
                            // Second pole position and field
                            R.z += tipsize;
                            r = norm(R);
                            B -= R * prefactor/(r*r*r);
                            
                            // Energy (B.M) * V
                            E[i + 1] = dot(B, m) * V;
                        }

                        // dF/dz = d²E/dz²
                        dFdz += ((E[0] - E[1]) + (E[2] - E[1])) / (delta*delta);
                    }
                }
            }
        }
    }
    kernel.setValueInCell(idx, 0, dFdz);
}

MFM::MFM(Magnet* magnet,
         const Grid grid)
    : grid_(grid),
      system_(std::make_shared<System>(magnet->world(), grid_)),
      lift(10e-9),
      tipsize(1e-3) {
    magnets_[magnet->name()] = magnet;
    if (grid_.size().z > 1) {
        throw std::invalid_argument("MFM should scan a 2D surface. Reduce"
                                    "the number of z-cells to 1.");
    }

    if (magnet->world()->pbcRepetitions().z > 0) {
        throw std::invalid_argument("Cannot take MFM picture of PBC in the"
                                    "z-direction.");
    }

}

MFM::MFM(const MumaxWorld* world,
         const Grid grid)
    : magnets_(world->magnets()),
      grid_(grid),
      system_(std::make_shared<System>(world, grid_)),
      lift(10e-9),
      tipsize(1e-3) {   
    if (grid_.size().z > 1) {
        throw std::invalid_argument("MFM should scan a 2D surface. Reduce the"
                                    "number of cells in the z direction to 1.");
    }

    if (world->pbcRepetitions().z > 0) {
        throw std::invalid_argument("MFM picture cannot be taken if PBC are"
                                    "enabled in the z-direction");
    }

}

Field MFM::eval() const {
    checkGridCompatibility();
    Field mfmTotal(system_, 1, 0.0);
    
    // loop over all magnets
    for (const auto& pair : magnets_) {
        Magnet* magnet = pair.second;

        Field mfm(system_, 1, 0.0);
        Grid mastergrid = magnet->world()->mastergrid();
        int3 pbcRepetitions = magnet->world()->pbcRepetitions();
        const real V = magnet->world()->cellVolume();
        int ncells = grid_.ncells();
        Field magnetization;
        if (const Ferromagnet* mag = magnet->asFM()) {
            magnetization = fullMagnetizationQuantity(mag).eval();
        } else if (const Antiferromagnet* mag = magnet->asAFM()) {
            magnetization = fullMagnetizationQuantity(mag).eval();
        } else {
            throw std::invalid_argument("Cannot calculate MFM of instance which"
                                        "is no Ferromagnet or Antiferromagnet.");
        }

        hipLaunchByPtr(ncells, k_magneticForceMicroscopy, mfm.cu(), magnetization.cu(), mastergrid, pbcRepetitions, lift, tipsize, V);
        mfmTotal += mfm;
    }
    return mfmTotal;
}

int MFM::ncomp() const {
    return 1;
}

std::shared_ptr<const System> MFM::system() const {
    return system_;
}

void MFM::checkGridCompatibility() const {
    // first, check if the xy-plane overlaps
    for (const auto& pair : magnets_) {
        Magnet* magnet = pair.second;
        int x1 = max(grid_.origin().x, magnet->system()->grid().origin().x);
        int y1 = max(grid_.origin().y, magnet->system()->grid().origin().y);
        int x2 = min(grid_.origin().x + grid_.size().x, magnet->system()->grid().origin().x + magnet->system()->grid().size().x);
        int y2 = min(grid_.origin().y + grid_.size().y, magnet->system()->grid().origin().y + magnet->system()->grid().size().y);
        if ((x2 - x1) > 0 && (y2 - y1) > 0) {
            // check if the height of the magnet intersects with the tip
            if (grid_.origin().z * magnet->world()->cellsize().z + lift - 1e-9 <= (magnet->grid().origin().z + magnet->grid().size().z) * magnet->world()->cellsize().z - magnet->world()->cellsize().z /2) {
                throw std::invalid_argument("Tip crashed into the sample. increase"
                                            "the lift or the origin of the MFM grid.");
            }
        }
    }
}