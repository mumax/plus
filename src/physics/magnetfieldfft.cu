#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include <memory>
#include <vector>

#include "constants.hpp"
#include "cudalaunch.hpp"
#include "field.hpp"
#include "magnetfieldfft.hpp"
#include "magnetfieldkernel.hpp"
#include "parameter.hpp"
#include "system.hpp"

#if FP_PRECISION == SINGLE
const hipfftType FFT = HIPFFT_R2C;
const hipfftType IFFT = HIPFFT_C2R;
const auto& fftExec = hipfftExecR2C;
const auto& ifftExec = hipfftExecC2R;
#elif FP_PRECISION == DOUBLE
const hipfftType FFT = HIPFFT_D2Z;
const hipfftType IFFT = HIPFFT_Z2D;
const auto& fftExec = hipfftExecD2Z;
const auto& ifftExec = hipfftExecZ2D;
#endif

#define __CUDAOP__ inline __device__ __host__

__CUDAOP__ complex operator+(complex a, complex b) {
#if FP_PRECISION == SINGLE
  return hipCaddf(a, b);
#elif FP_PRECISION == DOUBLE
  return hipCadd(a, b);
#endif
}

__CUDAOP__ complex operator*(complex a, complex b) {
#if FP_PRECISION == SINGLE
  return hipCmulf(a, b);
#elif FP_PRECISION == DOUBLE
  return hipCmul(a, b);
#endif
}

__global__ void k_pad(CuField out, CuField in, CuParameter msat) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (outIdx >= out.grid.ncells())
    return;

  int3 outCoo = out.grid.index2coord(outIdx);
  int3 inCoo = outCoo - out.grid.origin() + in.grid.origin();
  int inIdx = in.grid.coord2index(inCoo);

  if (in.grid.cellInGrid(inCoo)) {
    real Ms = msat.valueAt(inIdx);
    for (int c = 0; c < out.ncomp; c++)
      out.setValueInCell(outIdx, c, Ms * in.valueAt(inIdx, c));
  } else {
    for (int c = 0; c < out.ncomp; c++)
      out.setValueInCell(outIdx, c, 0.0);
  }
}

__global__ void k_unpad(CuField out, CuField in) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (outIdx >= out.grid.ncells())
    return;

  // Output coordinate relative to the origin of the output grid
  int3 outRelCoo = out.grid.index2coord(outIdx) - out.grid.origin();

  // Input coordinate relative to the origin of the input grid
  int3 inRelCoo = in.grid.size() - out.grid.size() + outRelCoo;

  int inIdx = in.grid.coord2index(inRelCoo + in.grid.origin());

  for (int c = 0; c < out.ncomp; c++) {
    out.setValueInCell(outIdx, c, in.valueAt(inIdx, c));
  }
}

static void checkCufftResult(hipfftResult result) {
  if (result != HIPFFT_SUCCESS)
    throw std::runtime_error("cufft error in demag convolution");
}

__global__ void k_apply_kernel_3d(complex* hx,
                                  complex* hy,
                                  complex* hz,
                                  complex* mx,
                                  complex* my,
                                  complex* mz,
                                  complex* kxx,
                                  complex* kyy,
                                  complex* kzz,
                                  complex* kxy,
                                  complex* kxz,
                                  complex* kyz,
                                  complex preFactor,
                                  int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  hx[i] = preFactor * (kxx[i] * mx[i] + kxy[i] * my[i] + kxz[i] * mz[i]);
  hy[i] = preFactor * (kxy[i] * mx[i] + kyy[i] * my[i] + kyz[i] * mz[i]);
  hz[i] = preFactor * (kxz[i] * mx[i] + kyz[i] * my[i] + kzz[i] * mz[i]);
}

__global__ void k_apply_kernel_2d(complex* hx,
                                  complex* hy,
                                  complex* hz,
                                  complex* mx,
                                  complex* my,
                                  complex* mz,
                                  complex* kxx,
                                  complex* kyy,
                                  complex* kzz,
                                  complex* kxy,
                                  complex preFactor,
                                  int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  hx[i] = preFactor * (kxx[i] * mx[i] + kxy[i] * my[i]);
  hy[i] = preFactor * (kxy[i] * mx[i] + kyy[i] * my[i]);
  hz[i] = preFactor * (kzz[i] * mz[i]);
}

MagnetFieldFFTExecutor::MagnetFieldFFTExecutor(Grid gridOut,
                                               Grid gridIn,
                                               real3 cellsize)
    : kernel_(gridOut, gridIn, cellsize), kfft(6), hfft(3), mfft(3) {
  int3 size = kernel_.grid().size();
  fftSize = {size.x / 2 + 1, size.y, size.z};
  int ncells = fftSize.x * fftSize.y * fftSize.z;

  for (auto& p : kfft)
    hipMalloc((void**)&p, ncells * sizeof(complex));
  for (auto& p : mfft)
    hipMalloc((void**)&p, ncells * sizeof(complex));
  for (auto& p : hfft)
    hipMalloc((void**)&p, ncells * sizeof(complex));

  checkCufftResult(hipfftPlan3d(&forwardPlan, size.z, size.y, size.x, FFT));
  checkCufftResult(hipfftPlan3d(&backwardPlan, size.z, size.y, size.x, IFFT));

  hipfftSetStream(forwardPlan, getCudaStream());
  hipfftSetStream(backwardPlan, getCudaStream());

  for (int comp = 0; comp < 6; comp++)
    checkCufftResult(
        fftExec(forwardPlan, kernel_.field().devptr(comp), kfft.at(comp)));
}

MagnetFieldFFTExecutor::~MagnetFieldFFTExecutor() {
  for (auto p : mfft)
    hipFree(p);
  for (auto p : kfft)
    hipFree(p);
  for (auto p : hfft)
    hipFree(p);

  checkCufftResult(hipfftDestroy(forwardPlan));
  checkCufftResult(hipfftDestroy(backwardPlan));
}

void MagnetFieldFFTExecutor::exec(Field* h,
                                  const Field* m,
                                  const Parameter* msat) const {
  // pad m, and multiply with msat
  System kernelSystem(h->system()->world(), kernel_.grid());
  std::unique_ptr<Field> mpad(new Field(&kernelSystem, 3));
  hipLaunchByPtr(mpad->grid().ncells(), k_pad, mpad->cu(), m->cu(), msat->cu());

  // Forward fourier transforms
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(fftExec(forwardPlan, mpad->devptr(comp), mfft.at(comp)));

  // apply kernel on m_fft
  int ncells = fftSize.x * fftSize.y * fftSize.z;
  complex preFactor{-MU0 / kernel_.grid().ncells(), 0};
  if (kernel_.grid().size().z == 1 && kernel_.grid().origin().z == 0) {
    // if the h field and m field are two dimensional AND are in the same plane
    // (kernel grid origin at z=0) then the kernel matrix has only 4 relevant
    // components and a more efficient cuda kernel can be used:
    hipLaunchByPtr(ncells, k_apply_kernel_2d, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), preFactor, ncells);
  } else {
    hipLaunchByPtr(ncells, k_apply_kernel_3d, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), kfft.at(4), kfft.at(5), preFactor,
               ncells);
  }

  // backward fourier transfrom
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(ifftExec(backwardPlan, hfft.at(comp), mpad->devptr(comp)));

  // unpad
  hipLaunchByPtr(h->grid().ncells(), k_unpad, h->cu(), mpad->cu());
}