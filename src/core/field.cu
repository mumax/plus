#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <memory>
#include <stdexcept>

#include "cudaerror.hpp"
#include "cudalaunch.hpp"
#include "cudastream.hpp"
#include "field.hpp"
#include "fieldops.hpp"
#include "fieldquantity.hpp"
#include "gpubuffer.hpp"
#include "system.hpp"

Field::Field() : system_(nullptr), ncomp_(0) {}

Field::Field(std::shared_ptr<const System> system, int nComponents)
    : system_(system), ncomp_(nComponents) {
  allocate();
}

Field::Field(std::shared_ptr<const System> system, int nComponents, real value)
    : Field(system, nComponents) {
  for (int comp = 0; comp < nComponents; comp++)
    setUniformComponent(comp, value);
}

Field::Field(const Field& other)
    : system_(other.system_), ncomp_(other.ncomp_) {
  buffers_ = other.buffers_;
  updateDevicePointersBuffer();
}

Field::Field(Field&& other) : system_(other.system_), ncomp_(other.ncomp_) {
  buffers_ = std::move(other.buffers_);
  bufferPtrs_ = std::move(other.bufferPtrs_);
  other.clear();
}

Field& Field::operator=(const Field& other) {
  if (this == &other)
    return *this;
  return *this = std::move(Field(other));  // moves a copy of other to this
}

Field& Field::operator=(const FieldQuantity& q) {
  return *this = std::move(q.eval());
}

Field& Field::operator=(Field&& other) {
  system_ = other.system_;
  ncomp_ = other.ncomp_;
  buffers_ = std::move(other.buffers_);
  bufferPtrs_ = std::move(other.bufferPtrs_);
  other.clear();
  return *this;
}

void Field::clear() {
  system_ = nullptr;
  ncomp_ = 0;
  free();
}

std::shared_ptr<const System> Field::system() const {
  return system_;
}

void Field::updateDevicePointersBuffer() {
  std::vector<real*> bufferPtrsOnHost(ncomp_);
  std::transform(buffers_.begin(), buffers_.end(), bufferPtrsOnHost.begin(),
                 [](auto& buf) { return buf.get(); });
  bufferPtrs_ = GpuBuffer<real*>(bufferPtrsOnHost);
}

void Field::allocate() {
  free();

  if (empty())
    return;

  buffers_ =
      std::vector<GpuBuffer<real>>(ncomp_, GpuBuffer<real>(grid().ncells()));

  updateDevicePointersBuffer();
}

void Field::free() {
  buffers_.clear();
  bufferPtrs_.recycle();
}

CuField Field::cu() const {
  return CuField(grid(), ncomp_, bufferPtrs_.get());
}

void Field::getData(real* buffer) const {
  for (int c = 0; c < ncomp_; c++) {
    real* bufferComponent = buffer + c * grid().ncells();
    checkCudaError(hipMemcpyAsync(bufferComponent, buffers_[c].get(),
                                   grid().ncells() * sizeof(real),
                                   hipMemcpyDeviceToHost, getCudaStream()));
  }
}

void Field::setData(real* buffer) {
  for (int c = 0; c < ncomp_; c++) {
    real* bufferComponent = buffer + c * grid().ncells();
    checkCudaError(hipMemcpyAsync(buffers_[c].get(), bufferComponent,
                                   grid().ncells() * sizeof(real),
                                   hipMemcpyHostToDevice, getCudaStream()));
  }
}

__global__ void k_setComponent(CuField f, real value, int comp) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (!f.cellInGrid(idx))
    return;
  f.setValueInCell(idx, comp, value);
}

void Field::setUniformComponent(int comp, real value) {
  hipLaunchByPtr(grid().ncells(), k_setComponent, cu(), value, comp);
}

void Field::makeZero() {
  for (int comp = 0; comp < ncomp_; comp++)
    setUniformComponent(comp, 0.0);
}

Field& Field::operator+=(const Field& other) {
  addTo(*this, 1, other);
  return *this;
}

Field& Field::operator-=(const Field& other) {
  addTo(*this, -1, other);
  return *this;
}

Field& Field::operator+=(const FieldQuantity& q) {
  addTo(*this, 1, q.eval());
  return *this;
}

Field& Field::operator-=(const FieldQuantity& q) {
  addTo(*this, -1, q.eval());
  return *this;
}
