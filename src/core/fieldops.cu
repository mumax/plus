#include "hip/hip_runtime.h"
#include <stdexcept>
#include <vector>

#include "cudalaunch.hpp"
#include "field.hpp"
#include "fieldops.hpp"
#include "reduce.hpp"

__global__ void k_addFields(CuField y,
                            real a1,
                            const CuField x1,
                            real a2,
                            const CuField x2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;
  for (int c = 0; c < y.ncomp; c++) {
    real term1 = a1 * x1.valueAt(idx, c % x1.ncomp);
    real term2 = a2 * x2.valueAt(idx, c % x2.ncomp);
    y.setValueInCell(idx, c, term1 + term2);
  }
}

__global__ void k_addFields(CuField y,
                            real3 a1,
                            const CuField x1,
                            real3 a2,
                            const CuField x2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;

  real3 term1 = a1 * x1.FM_vectorAt(idx);
  real3 term2 = a2 * x2.FM_vectorAt(idx);
  y.setVectorInCell(idx, term1 + term2);
}

__global__ void k_addFields(CuField y,
                            real6 a1,
                            const CuField x1,
                            real6 a2,
                            const CuField x2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;

  real6 term1 = a1 * x1.AFM_vectorAt(idx);
  real6 term2 = a2 * x2.AFM_vectorAt(idx);
  y.setVectorInCell(idx, term1 + term2);
}

inline void add(Field& y, real a1, const Field& x1, real a2, const Field& x2) {
  if (x1.system() != y.system() || x2.system() != y.system()) {
    throw std::invalid_argument(
        "Fields can not be added together because they belong to different "
        "systems)");
  }
  if ((x1.ncomp() != y.ncomp() || x1.ncomp() != y.ncomp()) ) {
    throw std::invalid_argument(
        "Fields can not be added because they do not have the same number of "
        "components");
  }
  int ncells = y.grid().ncells();
  hipLaunchByPtr(ncells, k_addFields, y.cu(), a1, x1.cu(), a2, x2.cu());
}

inline void add(Field& y,
                real3 a1,
                const Field& x1,
                real3 a2,
                const Field& x2) {
  if (x1.system() != y.system() || x2.system() != y.system()) {
    throw std::invalid_argument(
        "Fields can not be added together because they belong to different "
        "systems)");
  }
  if (x1.ncomp() != y.ncomp() || x1.ncomp() != y.ncomp()) {
    throw std::invalid_argument(
        "Fields can not be added because they do not have the same number of "
        "components");
  }
  if (x1.ncomp() != 3) {
    throw std::invalid_argument("Fields should have 3 components.");
  }
  int ncells = y.grid().ncells();
  hipLaunchByPtr(ncells, k_addFields, y.cu(), a1, x1.cu(), a2, x2.cu());
}

inline void add(Field& y,
                real6 a1,
                const Field& x1,
                real6 a2,
                const Field& x2) {
  if (x1.system() != y.system() || x2.system() != y.system()) {
    throw std::invalid_argument(
        "Fields can not be added together because they belong to different "
        "systems)");
  }
  if (x1.ncomp() != y.ncomp() || x1.ncomp() != y.ncomp()) {
    throw std::invalid_argument(
        "Fields can not be added because they do not have the same number of "
        "components");
  }
  if (x1.ncomp() != 6) {
    throw std::invalid_argument("Fields should have 6 components.");
  }
  int ncells = y.grid().ncells();
  hipLaunchByPtr(ncells, k_addFields, y.cu(), a1, x1.cu(), a2, x2.cu());
}

Field add(real a1, const Field& x1, real a2, const Field& x2) {
  Field y(x1.system(), x1.ncomp());
  add(y, a1, x1, a2, x2);
  return y;
}

Field add(const Field& x1, const Field& x2) {
  return add(1, x1, 1, x2);
}

void addTo(Field& y, real a, const Field& x) {
  add(y, 1, y, a, x);
}

void addTo(Field& y, real3 a, const Field& x) {
  real3 a0 = real3{1, 1, 1};
  add(y, a0, y, a, x);
}

void addTo(Field& y, real6 a, const Field& x) {
  real6 a0 = real6{1, 1, 1, 1, 1, 1};
  add(y, a0, y, a, x);
}

// TODO: this can be done much more efficient
Field add(std::vector<const Field*> x, std::vector<real> weights) {
  // TODO:: throw error if inputs are not compatible
  Field y = weights.at(0) * (*x.at(0));
  if (x.size() == 1) {
    return y;
  }

  for (int n = 1; n < x.size(); n++) {
    if (weights.at(n) != 0.0) {
      addTo(y, weights.at(n), *x.at(n));
    }
  }
  return y;
}

Field operator*(real a, const Field& x) {
  return add(0, x, a, x);
}

__global__ void k_addConstant(CuField y,
                              const CuField x,
                              real value,
                              int comp) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!y.cellInGeometry(idx))
    return;
  y.setValueInCell(idx, comp, x.valueAt(idx, comp) + value);
}

__global__ void k_normalize(CuField dst, const CuField src) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!dst.cellInGeometry(idx))
    return;

  int comp = src.ncomp;
  
  if (comp == 3) {
    real norm2 = 0.0;
    for (int c = 0; c < comp; c++) {
      real v = src.valueAt(idx, c);
      norm2 += v * v;
    }
    real invnorm = rsqrt(norm2);
    for (int c = 0; c < comp; c++) {
      real value = src.valueAt(idx, c) * invnorm;
      dst.setValueInCell(idx, c, value);
    }
  }
  else if (comp == 6) {
    real2 norm2 = real2{0., 0.};
    for (int c = 0; c < comp - 3; c++) {
      real v = src.valueAt(idx, c);
      real u = src.valueAt(idx, c + 3);
      norm2 += real2{v * v, u * u};
    }
    real2 invnorm = real2{rsqrt(norm2.x), rsqrt(norm2.y)};
    for (int c = 0; c < comp - 3; c++) {
      real vvalue = src.valueAt(idx, c) * invnorm.x;
      real uvalue = src.valueAt(idx, c + 3) * invnorm.y;
      dst.setValueInCell(idx, c, vvalue);
      dst.setValueInCell(idx, c + 3, uvalue);
    }
  }
}


Field normalized(const Field& src) {
  Field dst(Field(src.system(), src.ncomp()));
  hipLaunchByPtr(dst.grid().ncells(), k_normalize, dst.cu(), src.cu());
  return dst;
}

void normalize(Field& f) {
  hipLaunchByPtr(f.grid().ncells(), k_normalize, f.cu(), f.cu());
}

Field operator*(real3 a, const Field& x) {
  Field y(x.system(), x.ncomp());
  real3 a0 = real3{0, 0, 0};
  add(y, a0, x, a, x);
  return y;
}

Field operator*(real6 a, const Field& x) {
  Field y(x.system(), x.ncomp());
  real6 a0 = real6{0, 0, 0, 0, 0, 0};
  add(y, a0, x, a, x);
  return y;
}

// --------------------------------------------------
// fieldGetRGB

const float pi = 3.1415926535897931f;

/// Transform 3D vector with norm<=1 to its RGB representation
__device__ real3 getRGB(real3 vec) {
  // This function uses float arithmatic, as there is no need for
  // double precision colors.

  // HSL
  float H = atan2f(vec.y, vec.x);
  float S = norm(vec);
  float L = 0.5f + 0.5f * vec.z;

  // HSL to RGB
  float Hp = 3.f * H/pi;
  if (Hp < 0.f) {Hp += 6.f;}  // in [0, 6)
  else if (Hp >= 6.f) {Hp -= 6.f;}
  float C = (L<=0.5f) ? 2.f*L*S : 2.f*(1.f-L)*S;
  float X = C * (1.f - fabs(fmodf(Hp, 2.f) - 1.f));
  float m = L - C / 2.f;

  float R = m, G = m, B = m;
  if (Hp < 1.f) {
    R += C;
    G += X;
  } else if (Hp < 2.f) {
    R += X;
    G += C;
  } else if (Hp < 3.f) {
    G += C;
    B += X;
  } else if (Hp < 4.f) {
    G += X;
    B += C;
  } else if (Hp < 5.f) {
    R += X;
    B += C;
  } else {  // Hp < 6
    R += C;
    B += X;
  }

  // clip RGB values to be in [0,1]
  R = fminf(fmaxf(R, 0.f), 1.f);
  G = fminf(fmaxf(G, 0.f), 1.f);
  B = fminf(fmaxf(B, 0.f), 1.f);

  return real3{R, G, B};  // convert to real3 for Field
}

/// execute getRGB on both sublattice vectors
__device__ real6 getRGB(real6 vec) {
  real3 RGB1 = getRGB(real3{vec.x1, vec.y1, vec.z1});
  real3 RGB2 = getRGB(real3{vec.x2, vec.y2, vec.z2});
  return real6{RGB1.x, RGB1.y, RGB1.z, RGB2.x, RGB2.y, RGB2.z};
}

/// Map 3D vector field (with norm<=1) to RGB
__global__ void k_fieldGetRGB3(CuField dst, const CuField src) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!dst.cellInGeometry(idx)) {
    // not in geometry, so make grey instead
    dst.setVectorInCell(idx, real3{0.5, 0.5, 0.5});
  } else {
    dst.setVectorInCell(idx, getRGB(src.FM_vectorAt(idx)));
  }
}

/// Map 6D vector field (with norm<=1) to RGB
__global__ void k_fieldGetRGB6(CuField dst, const CuField src) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (!dst.cellInGeometry(idx)) {
    // not in geometry, so make grey instead
    dst.setVectorInCell(idx, real6{0.5, 0.5, 0.5, 0.5, 0.5, 0.5});
  } else {
    dst.setVectorInCell(idx, getRGB(src.AFM_vectorAt(idx)));
  }
}


Field fieldGetRGB(const Field& src) {
  if (src.ncomp() == 3) {  // 3D
    Field dst =  (1./maxVecNorm(src)) * src;  // rescale to make maximum norm 1
    hipLaunchByPtr(dst.grid().ncells(), k_fieldGetRGB3, dst.cu(), dst.cu());  // src is dst
    return dst;
  } else if (src.ncomp() == 6) {  // 6D
    Field dst =  (1./maxVecNorm(src)) * src;  // rescale to make maximum norm 1
    hipLaunchByPtr(dst.grid().ncells(), k_fieldGetRGB6, dst.cu(), dst.cu());  // src is dst
    return dst;
  } else {
    throw std::invalid_argument(
            "getRGB can only operate on vector fields with 3 or 6 components.");
  }
}
